#include "hip/hip_runtime.h"
#include <helios/shapes/sphere.h>
#include <hermes/numeric/numeric.h>
#include <hermes/numeric/e_float.h>
#include <helios/geometry/utils.h>
#include <helios/core/mem.h>

using namespace hermes;

namespace helios {

HERMES_DEVICE_CALLABLE Sphere::Sphere(real_t rad, real_t z0, real_t z1, real_t pm) : radius_{rad}, phi_max{pm} {
  zmin = Numbers::clamp(fminf(z0, z1), -radius_, radius_);
  radius_ = rad;
  zmax = Numbers::clamp(fmaxf(z0, z1), -radius_, radius_);
  theta_min = acosf(Numbers::clamp<real_t>(zmin / radius_, -1, 1));
  theta_max = acosf(Numbers::clamp<real_t>(zmin / radius_, -1, 1));
  theta_max = Trigonometry::degrees2radians(Numbers::clamp<real_t>(pm, 0, 360));
}

HERMES_DEVICE_CALLABLE bbox3 Sphere::objectBound() const {
  // TODO use phi_max to compute a tighter bound
  return bbox3(point3(-radius_, -radius_, zmin), point3(radius_, radius_, zmax));
}

HERMES_DEVICE_CALLABLE QuadricIntersectionReturn Sphere::intersectQuadric(const Shape *shape,
                                                                          const Ray &r,
                                                                          real_t t_max) const {
  real_t phi;
  hermes::point3 pHit;
  // Transform _Ray_ origin and direction to object space
  hermes::point3i oi = transform(shape->w2o, hermes::point3i(r.o.x, r.o.y, r.o.z));
  hermes::vec3i di = transform(shape->w2o, hermes::vec3i(r.d.x, r.d.y, r.d.z));

  // Solve quadratic equation to compute sphere _t0_ and _t1_
  hermes::Interval<real_t> t0, t1;
  // Compute sphere quadratic coefficients
  hermes::Interval<real_t> a = di.x.sqr() + di.y.sqr() + di.z.sqr();
  hermes::Interval<real_t> b = 2.f * (di.x * oi.x + di.y * oi.y + di.z * oi.z);
  hermes::Interval<real_t> c = oi.x.sqr() + oi.y.sqr() + oi.z.sqr() - hermes::Interval<real_t>(radius_).sqr();

  // Compute sphere quadratic discriminant _discrim_
  hermes::vec3i v(oi - b / (2.f * a) * di);
  hermes::Interval<real_t> length = v.length();
  hermes::Interval<real_t> discrim =
      4.f * a * (hermes::Interval<real_t>(radius_) + length) * (hermes::Interval<real_t>(radius_) - length);
  if (discrim.low < 0)
    return {};

  // Compute quadratic $t$ values
  hermes::Interval<real_t> rootDiscrim = discrim.sqrt();
  hermes::Interval<real_t> q;
  if ((real_t) b < 0)
    q = -.5f * (b - rootDiscrim);
  else
    q = -.5f * (b + rootDiscrim);
  t0 = q / a;
  t1 = c / q;
  // Swap quadratic $t$ values so that _t0_ is the lesser
  if (t0.low > t1.low)
    hermes::Numbers::swap(t0, t1);

  // Check quadric shape _t0_ and _t1_ for nearest intersection
  if (t0.high > t_max || t1.low <= 0)
    return {};
  hermes::Interval<real_t> tShapeHit = t0;
  if (tShapeHit.low <= 0) {
    tShapeHit = t1;
    if (tShapeHit.high > t_max)
      return {};
  }

  // Compute sphere hit position and $\phi$
  pHit = hermes::point3(oi) + (real_t) tShapeHit * hermes::vec3(di);
  // Refine sphere intersection point
  pHit *= radius_ / hermes::distance(pHit, hermes::point3(0, 0, 0));

  if (pHit.x == 0 && pHit.y == 0)
    pHit.x = 1e-5f * radius_;
  phi = std::atan2(pHit.y, pHit.x);
  if (phi < 0)
    phi += 2 * hermes::Constants::pi;

  // Test sphere intersection against clipping parameters
  if ((zmin > -radius_ && pHit.z < zmin) || (zmax < radius_ && pHit.z > zmax) ||
      phi > phi_max) {
    if (tShapeHit == t1)
      return {};
    if (t1.high > t_max)
      return {};
    tShapeHit = t1;
    // Compute sphere hit position and $\phi$
    pHit = hermes::point3(oi) + (real_t) tShapeHit * hermes::vec3(di);
    // Refine sphere intersection point
    pHit *= radius_ / hermes::distance(pHit, hermes::point3(0, 0, 0));

    if (pHit.x == 0 && pHit.y == 0)
      pHit.x = 1e-5f * radius_;
    phi = std::atan2(pHit.y, pHit.x);
    if (phi < 0)
      phi += 2 * hermes::Constants::pi;

    if ((zmin > -radius_ && pHit.z < zmin) || (zmax < radius_ && pHit.z > zmax) ||
        phi > phi_max)
      return {};
  }

  // Return _QuadricIntersection_ for sphere intersection
  return QuadricIntersection{real_t(tShapeHit), pHit, phi};
}

HERMES_DEVICE_CALLABLE SurfaceInteraction Sphere::interactionFromIntersection(const Shape *shape,
                                                                              const QuadricIntersection &isect,
                                                                              hermes::vec3 wo,
                                                                              real_t time) const {
  hermes::point3 pHit = isect.p_obj;
  real_t phi = isect.phi;
  // Find parametric representation of sphere hit
  real_t u = phi / phi_max;
  real_t cosTheta = pHit.z / radius_;
  real_t theta = hermes::Trigonometry::safe_acos(cosTheta);
  real_t v = (theta - theta_min) / (theta_max - theta_min);
  // Compute sphere $\dpdu$ and $\dpdv$
  real_t zRadius = std::sqrt(hermes::Numbers::sqr(pHit.x) + hermes::Numbers::sqr(pHit.y));
  real_t cosPhi = pHit.x / zRadius, sinPhi = pHit.y / zRadius;
  hermes::vec3 dpdu(-phi_max * pHit.y, phi_max * pHit.x, 0);
  real_t sinTheta = hermes::Numbers::safe_sqrt(1 - hermes::Numbers::sqr(cosTheta));
  hermes::vec3 dpdv = (theta_max - theta_min) *
      hermes::vec3(pHit.z * cosPhi, pHit.z * sinPhi, -radius_ * sinTheta);

  // Compute sphere $\dndu$ and $\dndv$
  hermes::vec3 d2Pduu = -phi_max * phi_max * hermes::vec3(pHit.x, pHit.y, 0);
  hermes::vec3 d2Pduv =
      (theta_max - theta_min) * pHit.z * phi_max * hermes::vec3(-sinPhi, cosPhi, 0.);
  hermes::vec3 d2Pdvv = -hermes::Numbers::sqr(theta_max - theta_min) * hermes::vec3(pHit.x, pHit.y, pHit.z);
  // Compute coefficients for fundamental forms
  real_t E = hermes::dot(dpdu, dpdu), F = hermes::dot(dpdu, dpdv), G = hermes::dot(dpdv, dpdv);
  hermes::vec3 n = hermes::normalize(hermes::cross(dpdu, dpdv));
  real_t e = hermes::dot(n, d2Pduu), f = hermes::dot(n, d2Pduv), g = hermes::dot(n, d2Pdvv);

  // Compute $\dndu$ and $\dndv$ from fundamental form coefficients
  real_t EGF2 = hermes::Numbers::differenceOfProducts(E, G, F, F);
  real_t invEGF2 = (EGF2 == 0) ? real_t(0) : 1 / EGF2;
  hermes::normal3 dndu =
      hermes::normal3((f * F - e * G) * invEGF2 * dpdu + (e * F - f * E) * invEGF2 * dpdv);
  hermes::normal3 dndv =
      hermes::normal3((g * F - f * G) * invEGF2 * dpdu + (f * F - g * E) * invEGF2 * dpdv);

  // Compute error bounds for sphere intersection
  hermes::vec3 pError = hermes::Numbers::gamma(5) * hermes::abs((hermes::vec3) pHit);

  // Return _SurfaceInteraction_ for quadric intersection
  bool flipNormal = HELIOS_MASK_BIT(shape->flags, shape_flags::REVERSE_ORIENTATION) ^
      HELIOS_MASK_BIT(shape->flags, shape_flags::TRANSFORM_SWAP_HANDEDNESS);
  hermes::vec3 woObject = shape->w2o(wo);
  return transform(shape->o2w, SurfaceInteraction(
      hermes::point3i(pHit, pError),
      hermes::point2(u, v), woObject, dpdu, dpdv,
      dndu, dndv, time, flipNormal));
}

HERMES_DEVICE_CALLABLE ShapeIntersectionReturn Sphere::intersect(const Shape *shape, const Ray &r, real_t t_max) const {
  auto isect = intersectQuadric(shape, r, t_max);
  if (!isect)
    return {};
  auto intr = interactionFromIntersection(shape, *isect, -r.d, r.time);
  return ShapeIntersection{intr, isect->t_hit};

  real_t phi;
  point3 phit;
  // transform HRay to object space
  vec3f oErr, dErr;
  Ray ray = transform(shape->w2o, r, oErr, dErr);
  //    initialize efloat ray coordinate valyes
  EFloat ox(ray.o.x, oErr.x), oy(ray.o.y, oErr.y), oz(ray.o.z, oErr.z);
  EFloat dx(ray.d.x, dErr.x), dy(ray.d.y, dErr.y), dz(ray.d.z, dErr.z);
  // compute quadritic Sphere coefficients
  EFloat a = dx * dx + dy * dy + dz * dz;
  EFloat b = 2 * (dx * ox + dy * oy + dz * oz);
  EFloat c = ox * ox + oy * oy + oz * oz - EFloat(radius_) * EFloat(radius_);
  // solve quadritic equation for t values
  EFloat t0, t1;
  if (!solve_quadratic(a, b, c, &t0, &t1))
    return {};
  // check quadric shape t0 and t1 for nearest intersection
  if (t0.upperBound() > ray.max_t || t1.lowerBound() <= 0)
    return {};
  EFloat thit = t0;
  if (thit.lowerBound() <= 0) {
    thit = t1;
    if (thit.upperBound() > ray.max_t)
      return {};
  }
  // compute Sphere hit position and phi
  phit = ray((real_t) thit);
  // refine sphere intersection point
  phit *= radius_ / distance(phit, point3());
  if (phit.x == 0 && phit.y == 0)
    phit.x = 1e-5f * radius_;
  phi = atan2(phit.y, phit.x);
  if (phi < 0.)
    phi += 2 * Constants::pi;
  // test Sphere intersection against clipping parameters
  if ((zmin > -radius_ && phit.z < zmin) || (zmax < radius_ && phit.z > zmax) ||
      phi > phi_max) {
    if (thit == t1)
      return {};
    if (t1.upperBound() > ray.max_t)
      return {};
    thit = t1;
    // compute Sphere hit position and phi
    phit = ray((real_t) thit);
    // refine sphere intersection point
    phit *= radius_ / distance(phit, point3());
    if (phit.x == 0 && phit.y == 0)
      phit.x = 1e-5f * radius_;
    phi = atan2(phit.y, phit.x);
    if (phi < 0.)
      phi += 2 * Constants::pi;
    if ((zmin > -radius_ && phit.z < zmin) || (zmax < radius_ && phit.z > zmax) ||
        phi > phi_max)
      return {};
  }
  // find parametric representation of Sphere hit
  real_t u = phi / phi_max;
  real_t theta = acosf(Numbers::clamp<real_t>(phit.z / radius_, -1, 1));
  real_t v = (theta - theta_min) / (theta_max - theta_min);
  // compute Sphere dp/du and dp/dv
  real_t zradius = sqrt(phit.x * phit.x + phit.y * phit.y);
  real_t invradius = 1.f / zradius;
  real_t cosphi = phit.x * invradius;
  real_t sinphi = phit.y * invradius;
  vec3f dpdu(-phi_max * phit.y, phi_max * phit.x, 0);
  vec3f dpdv = (theta_max - theta_min) * vec3(phit.z * cosphi, phit.z * sinphi,
                                              -radius_ * sin(theta));
  // compute Sphere dn/du and dn/dv
  vec3 d2Pduu = -phi_max * phi_max * vec3f(phit.x, phit.y, 0);
  vec3 d2Pduv =
      (theta_max - theta_min) * phit.z * phi_max * vec3f(-sinphi, cosphi, 0.f);
  vec3f d2Pdvv = -(theta_max - theta_min) * (theta_max - theta_min) *
      vec3f(phit.x, phit.y, phit.z);
  // compute coefficients for fundamental forms
  real_t E = dot(dpdu, dpdu);
  real_t F = dot(dpdu, dpdv);
  real_t G = dot(dpdv, dpdv);
  vec3f N = normalize(cross(dpdu, dpdv));
  real_t e = dot(N, d2Pduu);
  real_t f = dot(N, d2Pduv);
  real_t g = dot(N, d2Pdvv);
  // compute dndu and dndv from fundamental form coefficients
  real_t invEFG2 = 1 / (E * G - F * F);
  normal3f dndu((f * F - e * G) * invEFG2 * dpdu +
      (e * F - f * E) * invEFG2 * dpdv);
  normal3f dndv((g * F - f * G) * invEFG2 * dpdu +
      (f * F - g * E) * invEFG2 * dpdv);
  // compute error bounds for sphere intersection
  vec3f pError = Numbers::gamma(5) * abs((vec3f) phit);

  return ShapeIntersection{
      .interaction = transform(shape->o2w, SurfaceInteraction(phit, pError, point2f(u, v),
                                                              -ray.d, dpdu, dpdv, dndu, dndv,
                                                              ray.time)),
      .t_hit = (real_t) (thit)
  };
}

HERMES_DEVICE_CALLABLE bool Sphere::intersectP(const Shape *shape, const Ray &r, real_t t_max) const {
  return intersectQuadric(shape, r).hasValue();
  real_t phi;
  point3 phit;
  // transform HRay to object space
  vec3f oErr, dErr;
  Ray ray = transform(shape->w2o, r, oErr, dErr);
  //    initialize efloat ray coordinate valyes
  EFloat ox(ray.o.x, oErr.x), oy(ray.o.y, oErr.y), oz(ray.o.z, oErr.z);
  EFloat dx(ray.d.x, dErr.x), dy(ray.d.y, dErr.y), dz(ray.d.z, dErr.z);
  // compute quadratic Sphere coefficients
  EFloat a = dx * dx + dy * dy + dz * dz;
  EFloat b = 2 * (dx * ox + dy * oy + dz * oz);
  EFloat c = ox * ox + oy * oy + oz * oz - EFloat(radius_) * EFloat(radius_);
  // solve quadritic equation for t values
  EFloat t0, t1;
  if (!solve_quadratic(a, b, c, &t0, &t1))
    return false;
  HERMES_LOG_VARIABLE((float) t0.lowerBound())
  HERMES_LOG_VARIABLE((float) t0)
  HERMES_LOG_VARIABLE((float) t0.upperBound())
  HERMES_LOG_VARIABLE((float) t1.lowerBound())
  HERMES_LOG_VARIABLE((float) t1)
  HERMES_LOG_VARIABLE((float) t1.upperBound())
  // check quadric shape t0 and t1 for nearest intersection
  if (t0.upperBound() > ray.max_t || t1.lowerBound() <= 0)
    return false;
  EFloat thit = t0;
  if (thit.lowerBound() <= 0) {
    thit = t1;
    HERMES_LOG_VARIABLE((float) thit)
    HERMES_LOG_VARIABLE((float) thit.upperBound())
    HERMES_LOG_VARIABLE((float) thit.lowerBound())
    if (thit.upperBound() > ray.max_t)
      return false;
  }
  HERMES_LOG_VARIABLE((float) thit)
  // compute Sphere hit position and phi
  phit = ray((real_t) thit);
  // refine sphere intersection point
  phit *= radius_ / distance(phit, point3());
  if (phit.x == 0 && phit.y == 0)
    phit.x = 1e-5f * radius_;
  phi = atan2(phit.y, phit.x);
  if (phi < 0.)
    phi += 2 * Constants::pi;
  // test Sphere intersection against clipping parameters
  if ((zmin > -radius_ && phit.z < zmin) || (zmax < radius_ && phit.z > zmax) ||
      phi > phi_max) {
    if (thit == t1)
      return false;
    if (t1.upperBound() > ray.max_t)
      return false;
    thit = t1;
    // compute Sphere hit position and phi
    phit = ray((real_t) thit);
    // refine sphere intersection point
    phit *= radius_ / distance(phit, point3());
    if (phit.x == 0 && phit.y == 0)
      phit.x = 1e-5f * radius_;
    phi = atan2(phit.y, phit.x);
    if (phi < 0.)
      phi += 2 * Constants::pi;
    if ((zmin > -radius_ && phit.z < zmin) || (zmax < radius_ && phit.z > zmax) ||
        phi > phi_max)
      return false;
  }

  return true;
}

HERMES_DEVICE_CALLABLE real_t Sphere::surfaceArea() const { return phi_max * radius_ * (zmax - zmin); }

HERMES_DEVICE_CALLABLE real_t Sphere::radius() const {
  return radius_;
}

} // namespace helios
