#include "hip/hip_runtime.h"
#include <helios/shapes/sphere.h>
#include <hermes/numeric/numeric.h>
#include <hermes/numeric/e_float.h>
#include <helios/geometry/utils.h>

using namespace hermes;

namespace helios {

Shape Sphere::createShape(const Sphere &sphere, const hermes::Transform &o2w) {
  return {
      .o2w = o2w,
      .w2o = inverse(o2w),
      .bounds = o2w(sphere.objectBound()),
      .primitive_data = nullptr,
      .type = ShapeType::SPHERE,
      .flags = shape_flags::NONE
  };
}

HERMES_DEVICE_CALLABLE Sphere::Sphere(real_t rad, real_t z0, real_t z1, real_t pm) : radius{rad}, phi_max{pm} {
  zmin = Numbers::clamp(fminf(z0, z1), -radius, radius);
  radius = rad;
  zmax = Numbers::clamp(fmaxf(z0, z1), -radius, radius);
  theta_min = acosf(Numbers::clamp<real_t>(zmin / radius, -1, 1));
  theta_max = acosf(Numbers::clamp<real_t>(zmin / radius, -1, 1));
  theta_max = Trigonometry::degrees2radians(Numbers::clamp<real_t>(pm, 0, 360));
}

HERMES_DEVICE_CALLABLE bbox3 Sphere::objectBound() const {
  // TODO use phi_max to compute a tighter bound
  return bbox3(point3(-radius, -radius, zmin), point3(radius, radius, zmax));
}

HERMES_DEVICE_CALLABLE bool Sphere::intersect(const Shape *shape, const Ray &r, real_t *tHit, SurfaceInteraction *isect,
                                              bool test_alpha_texture) const {
  real_t phi;
  point3 phit;
  // transform HRay to object space
  vec3f oErr, dErr;
  Ray ray = transform(shape->w2o, r, oErr, dErr);
  //    initialize efloat ray coordinate valyes
  EFloat ox(ray.o.x, oErr.x), oy(ray.o.y, oErr.y), oz(ray.o.z, oErr.z);
  EFloat dx(ray.d.x, dErr.x), dy(ray.d.y, dErr.y), dz(ray.d.z, dErr.z);
  // compute quadritic Sphere coefficients
  EFloat a = dx * dx + dy * dy + dz * dz;
  EFloat b = 2 * (dx * ox + dy * oy + dz * oz);
  EFloat c = ox * ox + oy * oy + oz * oz - EFloat(radius) * EFloat(radius);
  // solve quadritic equation for t values
  EFloat t0, t1;
  if (!solve_quadratic(a, b, c, &t0, &t1))
    return false;
  // check quadric shape t0 and t1 for nearest intersection
  if (t0.upperBound() > ray.max_t || t1.lowerBound() <= 0)
    return false;
  EFloat thit = t0;
  if (thit.lowerBound() <= 0) {
    thit = t1;
    if (thit.upperBound() > ray.max_t)
      return false;
  }
  // compute Sphere hit position and phi
  phit = ray((real_t) thit);
  // refine sphere intersection point
  phit *= radius / distance(phit, point3());
  if (phit.x == 0 && phit.y == 0)
    phit.x = 1e-5f * radius;
  phi = atan2(phit.y, phit.x);
  if (phi < 0.)
    phi += 2 * Constants::pi;
  // test Sphere intersection against clipping parameters
  if ((zmin > -radius && phit.z < zmin) || (zmax < radius && phit.z > zmax) ||
      phi > phi_max) {
    if (thit == t1)
      return false;
    if (t1.upperBound() > ray.max_t)
      return false;
    thit = t1;
    // compute Sphere hit position and phi
    phit = ray((real_t) thit);
    // refine sphere intersection point
    phit *= radius / distance(phit, point3());
    if (phit.x == 0 && phit.y == 0)
      phit.x = 1e-5f * radius;
    phi = atan2(phit.y, phit.x);
    if (phi < 0.)
      phi += 2 * Constants::pi;
    if ((zmin > -radius && phit.z < zmin) || (zmax < radius && phit.z > zmax) ||
        phi > phi_max)
      return false;
  }
  // find parametric representation of Sphere hit
  real_t u = phi / phi_max;
  real_t theta = acosf(Numbers::clamp<real_t>(phit.z / radius, -1, 1));
  real_t v = (theta - theta_min) / (theta_max - theta_min);
  // compute Sphere dp/du and dp/dv
  real_t zradius = sqrt(phit.x * phit.x + phit.y * phit.y);
  real_t invradius = 1.f / zradius;
  real_t cosphi = phit.x * invradius;
  real_t sinphi = phit.y * invradius;
  vec3f dpdu(-phi_max * phit.y, phi_max * phit.x, 0);
  vec3f dpdv = (theta_max - theta_min) * vec3(phit.z * cosphi, phit.z * sinphi,
                                              -radius * sin(theta));
  // compute Sphere dn/du and dn/dv
  vec3 d2Pduu = -phi_max * phi_max * vec3f(phit.x, phit.y, 0);
  vec3 d2Pduv =
      (theta_max - theta_min) * phit.z * phi_max * vec3f(-sinphi, cosphi, 0.f);
  vec3f d2Pdvv = -(theta_max - theta_min) * (theta_max - theta_min) *
      vec3f(phit.x, phit.y, phit.z);
  // compute coefficients for fundamental forms
  real_t E = dot(dpdu, dpdu);
  real_t F = dot(dpdu, dpdv);
  real_t G = dot(dpdv, dpdv);
  vec3f N = normalize(cross(dpdu, dpdv));
  real_t e = dot(N, d2Pduu);
  real_t f = dot(N, d2Pduv);
  real_t g = dot(N, d2Pdvv);
  // compute dndu and dndv from fundamental form coefficients
  real_t invEFG2 = 1 / (E * G - F * F);
  normal3f dndu((f * F - e * G) * invEFG2 * dpdu +
      (e * F - f * E) * invEFG2 * dpdv);
  normal3f dndv((g * F - f * G) * invEFG2 * dpdu +
      (f * F - g * E) * invEFG2 * dpdv);
  // compute error bounds for sphere intersection
  vec3f pError = Numbers::gamma(5) * abs((vec3f) phit);
  // initialize SurfaceInteraction from parametric information
  *isect = transform(shape->o2w, SurfaceInteraction(phit, pError, point2f(u, v),
                                                   -ray.d, dpdu, dpdv, dndu, dndv,
                                                   ray.time, shape));
  // update tHit for quadric intersection
  *tHit = (real_t) (thit);
  return true;
}

HERMES_DEVICE_CALLABLE bool Sphere::intersectP(const Shape &shape, const Ray &r, bool test_alpha_texture) const {
  real_t phi;
  point3 phit;
  // transform HRay to object space
  vec3f oErr, dErr;
  Ray ray = transform(shape.w2o, r, oErr, dErr);
  //    initialize efloat ray coordinate valyes
  EFloat ox(ray.o.x, oErr.x), oy(ray.o.y, oErr.y), oz(ray.o.z, oErr.z);
  EFloat dx(ray.d.x, dErr.x), dy(ray.d.y, dErr.y), dz(ray.d.z, dErr.z);
  // compute quadritic Sphere coefficients
  EFloat a = dx * dx + dy * dy + dz * dz;
  EFloat b = 2 * (dx * ox + dy * oy + dz * oz);
  EFloat c = ox * ox + oy * oy + oz * oz - EFloat(radius) * EFloat(radius);
  // solve quadritic equation for t values
  EFloat t0, t1;
  if (!solve_quadratic(a, b, c, &t0, &t1))
    return false;
  // check quadric shape t0 and t1 for nearest intersection
  if (t0.upperBound() > ray.max_t || t1.lowerBound() <= 0)
    return false;
  EFloat thit = t0;
  if (thit.lowerBound() <= 0) {
    thit = t1;
    if (thit.upperBound() > ray.max_t)
      return false;
  }
  // compute Sphere hit position and phi
  phit = ray((real_t) thit);
  // refine sphere intersection point
  phit *= radius / distance(phit, point3());
  if (phit.x == 0 && phit.y == 0)
    phit.x = 1e-5f * radius;
  phi = atan2(phit.y, phit.x);
  if (phi < 0.)
    phi += 2 * Constants::pi;
  // test Sphere intersection against clipping parameters
  if ((zmin > -radius && phit.z < zmin) || (zmax < radius && phit.z > zmax) ||
      phi > phi_max) {
    if (thit == t1)
      return false;
    if (t1.upperBound() > ray.max_t)
      return false;
    thit = t1;
    // compute Sphere hit position and phi
    phit = ray((real_t) thit);
    // refine sphere intersection point
    phit *= radius / distance(phit, point3());
    if (phit.x == 0 && phit.y == 0)
      phit.x = 1e-5f * radius;
    phi = atan2(phit.y, phit.x);
    if (phi < 0.)
      phi += 2 * Constants::pi;
    if ((zmin > -radius && phit.z < zmin) || (zmax < radius && phit.z > zmax) ||
        phi > phi_max)
      return false;
  }

  return true;
}

HERMES_DEVICE_CALLABLE real_t Sphere::surfaceArea() const { return phi_max * radius * (zmax - zmin); }

} // namespace helios
