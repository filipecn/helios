#include "hip/hip_runtime.h"
#include <helios/samplers/stratified_sampler.h>

using namespace hermes;

namespace helios {

//void StratifiedSampler::latinHypercube(real_t *samples, int nSamples, int nDim, PCGRNG &rng) {
//   generate LHS samples along diagonal
//  real_t invNSamples = static_cast<real_t>(1) / nSamples;
//  for (int i = 0; i < nSamples; ++i)
//    for (int j = 0; j < nDim; ++j) {
//      real_t sj = (i + (rng.uniformFloat())) * invNSamples;
//    }
//   permute LHS samples in each dimension
//  for (int i = 0; i < nDim; ++i)
//    for (int j = 0; j < nSamples; ++j) {
//      int other = j + rng.uniformUInt32(nSamples - j);
//      std::swap(samples[nDim * j + i], samples[nDim * other + i]);
//    }
//}

StratifiedSampler::StratifiedSampler(const size2 &grid_resolution,
                                     bool jitter_samples,
                                     u32 sampled_dimensions)
    : resolution_(grid_resolution), jitter_samples_(jitter_samples) {
  SamplePool::PoolDescriptor descriptor = {
      .samples_per_pixel = resolution_.total(),
      .dimensions = sampled_dimensions,
      .array1_sizes = {},
      .array2_sizes = {}
  };
  pool_.set(descriptor);
}

HERMES_DEVICE_CALLABLE StratifiedSampler::StratifiedSampler(const StratifiedSampler &other) {
  resolution_ = other.resolution_;
  jitter_samples_ = other.jitter_samples_;
  pool_ = other.pool_;
  rng = other.rng;
}

HERMES_DEVICE_CALLABLE StratifiedSampler::~StratifiedSampler() {}

HERMES_DEVICE_CALLABLE void StratifiedSampler::startPixel(const index2 &p) {
  const u32 sampled_dimensions = pool_.dimensionCount();
  const real_t inv_n_samples = static_cast<real_t>(1) / resolution_.total();
  const real_t dx = static_cast<real_t>(1) / resolution_.width;
  const real_t dy = static_cast<real_t>(1) / resolution_.height;

  for (auto ij : range2(resolution_)) {
    u32 pixel_sample = ij.j * resolution_.width + ij.i;
    // generate samples for dimensions
    for (u32 d = 0; d < sampled_dimensions; ++d) {
      // generate 1d sample
      real_t delta = jitter_samples_ ? rng.uniformFloat() : 0.5f;
      pool_.get1DSample(pixel_sample, d) = fminf((pixel_sample + delta) * inv_n_samples, Constants::one_minus_epsilon);
      // generate 2d sample
      real_t jx = jitter_samples_ ? rng.uniformFloat() : 0.5f;
      real_t jy = jitter_samples_ ? rng.uniformFloat() : 0.5f;
      point2 &sample_p = pool_.get2DSample(pixel_sample, d);
      sample_p.x = fminf((ij.i + jx) * dx, Constants::one_minus_epsilon);
      sample_p.y = fminf((ij.j + jy) * dy, Constants::one_minus_epsilon);
    }
    // generate samples for 1d arrays
    for (u32 a = 0; a < pool_.array1Count(); ++a) {
      real_t *arr = pool_.get1DArray(pixel_sample, a);
      for (u32 j = 0; j < pool_.get1DArraySize(a); ++j) {
        real_t delta = jitter_samples_ ? rng.uniformFloat() : 0.5f;
        arr[j] = fminf((pixel_sample + delta) * inv_n_samples, Constants::one_minus_epsilon);
      }
    }
    for (u32 a = 0; a < pool_.array2Count(); ++a) {
      point2 *arr = pool_.get2DArray(pixel_sample, a);
      for (u32 j = 0; j < pool_.get2DArraySize(a); ++j) {
        real_t jx = jitter_samples_ ? rng.uniformFloat() : 0.5f;
        real_t jy = jitter_samples_ ? rng.uniformFloat() : 0.5f;
        arr[j].x = fminf((ij.i + jx) * dx, Constants::one_minus_epsilon);
        arr[j].y = fminf((ij.j + jy) * dy, Constants::one_minus_epsilon);
      }
    }
  }

  // shuffle samples
  pool_.reset();
}

HERMES_DEVICE_CALLABLE void StratifiedSampler::setDataPtr(byte *data) {
  pool_.setData(data);
}

HERMES_DEVICE_CALLABLE SamplePool::size_type StratifiedSampler::memorySize() const {
  return pool_.itemSize();
}

HERMES_DEVICE_CALLABLE SamplePool StratifiedSampler::samplePool() const {
  return pool_;
}

HERMES_DEVICE_CALLABLE CameraSample StratifiedSampler::cameraSample(const index2 &p) {
  CameraSample cs;
  cs.film = pool_.get2DSample() + vec2(p.i, p.j);
  cs.time = pool_.get1DSample();
  cs.lens = pool_.get2DSample();
  return cs;
}

HERMES_DEVICE_CALLABLE bool StratifiedSampler::startNextSample() {
  return pool_.startNextSample();
}

HERMES_DEVICE_CALLABLE void StratifiedSampler::setIndex(u32 i) {
  pool_.setPoolIndex(i);
}

HERMES_DEVICE_CALLABLE u32 StratifiedSampler::samplesPerPixel() const {
  return resolution_.total();
}

// StratifiedSampler::StratifiedSampler(int xstart, int xend, int ystart, int
// yend,
//                                      int xs, int ys, bool jitter, float
//                                      sopen, float sclose)
//     : Sampler(xstart, xend, ystart, yend, xs * ys, sopen, sclose) {
//   jitterSamples = jitter;
//   xPos = xPixelStart;
//   yPos = yPixelStart;
//   xPixelSamples = xs;
//   yPixelSamples = ys;
//   buffer = new float[5 * xPixelSamples * yPixelSamples];
// }

// int StratifiedSampler::getMoreSamples(Sample *samples, ponos::RNG &rng) {
//   if (yPos == yPixelEnd)
//     return 0;
//   int nSamples = xPixelSamples * yPixelSamples;
//   // Generate stratified camera samples for (xPos, yPos)
//   // // generate initial stratified samples into buffer memory
//   float *bufp = buffer;
//   float *imageSamples = bufp;
//   bufp += 2 * nSamples;
//   float *lensSamples = bufp;
//   bufp += 2 * nSamples;
//   float *timeSamples = bufp;
//   generateStratifiedSample2D(imageSamples, xPixelSamples, yPixelSamples, rng,
//                              jitterSamples);
//   generateStratifiedSample2D(imageSamples, xPixelSamples, yPixelSamples, rng,
//                              jitterSamples);
//   generateStratifiedSample1D(imageSamples, xPixelSamples * yPixelSamples,
//   rng,
//                              jitterSamples);
//   // // shift stratified image samples to pixel coordinates
//   for (int o = 0; o < 2 * xPixelSamples * yPixelSamples; o += 2) {
//     imageSamples[o] += xPos;
//     imageSamples[o + 1] += yPos;
//   }
//   // // decorrelate sample dimensions
//   // TODO pg 354 shuffle(lensSamples, xPixelSamples * yPixelSamples, 2, rng);
//   // TODO shuffle(timeSamples, xPixelSamples * yPixelSamples, 1, rng);
//   // // initialize stratified samples with sample values
//   for (int i = 0; i < nSamples; i++) {
//     samples[i].imageX = imageSamples[2 * i];
//     samples[i].imageY = imageSamples[2 * i + 1];
//     samples[i].lensU = lensSamples[2 * i];
//     samples[i].lensV = lensSamples[2 * i + 1];
//     samples[i].time = ponos::lerp(timeSamples[i], shutterOpen, shutterClose);
//     // generate stratified samples for integrators
//     for (uint32 j = 0; j < samples[i].n1D.size(); i++)
//       generateLatinHypercube(samples[i].oneD[j], samples[i].n1D[j], 1, rng);
//     for (uint32 j = 0; j < samples[i].n2D.size(); i++)
//       generateLatinHypercube(samples[i].twoD[j], samples[i].n2D[j], 2, rng);
//   }
//   // Advance to next pixel for stratified sampling
//   if (++xPos == xPixelSamples) {
//     xPos = xPixelStart;
//     ++yPos;
//   }
//   return nSamples;
// }

// Sampler *StratifiedSampler::getSubSampler(int num, int count) {
//   int x0, x1, y0, y1;
//   computeSubWindow(num, count, &x0, &x1, &y0, &y1);
//   if (x0 == x1 || y0 == y1)
//     return nullptr;
//   return new StratifiedSampler(x0, x1, y0, y1, xPixelSamples, yPixelSamples,
//                                jitterSamples, shutterOpen, shutterClose);
// }

} // namespace helios
