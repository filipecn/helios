#include "hip/hip_runtime.h"
#include <helios/geometry/ray.h>
#include <hermes/numeric/numeric.h>

using namespace hermes;

namespace helios {

HERMES_DEVICE_CALLABLE Ray::Ray() : max_t(hermes::Constants::real_infinity), time(0.f) {}

HERMES_DEVICE_CALLABLE Ray::Ray(const point3 &origin, const vec3 &direction, real_t tMax,
                                real_t time)
    : o(origin), d(direction), max_t(tMax), time(time) /*, medium(medium)*/ {}

HERMES_DEVICE_CALLABLE point3 Ray::operator()(real_t t) const { return o + d * t; }

hermes::MemoryDumper::RegionLayout Ray::memoryDumpLayout() {
  return MemoryDumper::RegionLayout().withSizeOf<Ray>()
      .withSubRegion(point3::memoryDumpLayout().withColor(ConsoleColors::red))
      .withSubRegion(vec3::memoryDumpLayout().withColor(ConsoleColors::green))
      .withSubRegion(
          MemoryDumper::RegionLayout()
              .withSizeOf<real_t>().withColor(ConsoleColors::blue).withTypeFrom<real_t>())
      .withSubRegion(
          MemoryDumper::RegionLayout()
              .withSizeOf<real_t>().withColor(ConsoleColors::magenta).withTypeFrom<real_t>());
}

HERMES_DEVICE_CALLABLE point3 Ray::offsetRayOrigin(const hermes::point3 &p, const hermes::vec3 &pError,
                                                   const hermes::normal3 &n, const hermes::vec3 &w) {
  real_t d = dot(abs(n), pError);
  vec3 offset = d * vec3(n);
  if (dot(w, n) != 0)
    offset = -offset;
  point3 po = p + offset;
  // round offset point away from p
  for (int i = 0; i < 3; i++)
    if (offset[i] < 0)
      po[i] = Numbers::nextFloatUp(po[i]);
    else if (offset[i] > 0)
      po[i] = Numbers::nextFloatDown(po[i]);
  return po;
}

HERMES_DEVICE_CALLABLE RayDifferential::RayDifferential() { has_differentials = false; }

HERMES_DEVICE_CALLABLE RayDifferential::RayDifferential(const point3 &origin, const vec3 &direction,
                                                        real_t tMax, real_t time)
    : ray(origin, direction, tMax, time) {
  has_differentials = false;
}

HERMES_DEVICE_CALLABLE void RayDifferential::scaleDifferentials(float s) {
  rx_origin = ray.o + (rx_origin - ray.o) * s;
  ry_origin = ray.o + (ry_origin - ray.o) * s;
  rx_direction = ray.d + (rx_direction - ray.d) * s;
  ry_direction = ray.d + (ry_direction - ray.d) * s;
}

hermes::MemoryDumper::RegionLayout RayDifferential::memoryDumpLayout() {
  return MemoryDumper::RegionLayout().withSizeOf<RayDifferential>()
      .withSubRegion(Ray::memoryDumpLayout())
      .withSubRegion(point3::memoryDumpLayout().withColor(ConsoleColors::cyan))
      .withSubRegion(point3::memoryDumpLayout().withColor(ConsoleColors::magenta))
      .withSubRegion(vec3::memoryDumpLayout().withColor(ConsoleColors::red))
      .withSubRegion(vec3::memoryDumpLayout().withColor(ConsoleColors::green))
      .withSubRegion(MemoryDumper::RegionLayout().withSizeOf<bool>().withTypeFrom<bool>());
}

} // namespace helios
