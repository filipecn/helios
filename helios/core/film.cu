#include "hip/hip_runtime.h"
#include <helios/core/film.h>
#include <helios/common/globals.h>
#include <hermes/common/cuda_utils.h>

#include <utility>

using namespace hermes;

namespace helios {

HERMES_DEVICE_CALLABLE FilmTile::FilmTile(const bounds2i &pixel_bounds, const PCF *filter)
    : pixel_bounds_(pixel_bounds), filter_(filter) {
  pixels_ = new FilmTilePixel[pixel_bounds.area()];
}

HERMES_DEVICE_CALLABLE FilmTile::~FilmTile() {
  delete[] pixels_;
}

HERMES_DEVICE_CALLABLE void FilmTile::addSample(const point2 &p, const SpectrumOld &L,
                                                real_t sample_weight) {
  // compute sample's raster bounds
  point2 p_discrete = p - vec2(0.5f);
  index2 p0 = ceil(p_discrete - filter_->radius);
  index2 p1 = floor(p_discrete + filter_->radius) + vec2(1);
  p0 = max(p0, pixel_bounds_.lower());
  p1 = min(p1, pixel_bounds_.upper());

//  printf("%d %d %d %d (%d %d %d %d)\n",
//         pixel_bounds_.lower().i,
//         pixel_bounds_.lower().j,
//         pixel_bounds_.upper().i,
//         pixel_bounds_.upper().j,
//         p0.i, p0.j, p1.i, p1.j);

  // loop over filter support and add sample to pixel arrays
  // precompute x and y filter table offsets
  int *ifx = new int[p1.i - p0.i];
  for (int x = p0.i; x < p1.i; ++x) {
    real_t fx =
        abs((x - p_discrete.x) * filter_->inv_radius.x * PCF::filter_table_width);
    ifx[x - p0.i] = min((int) floor(fx), PCF::filter_table_width - 1);
  }
  int *ify = new int[p1.j - p0.j];
  for (int y = p0.j; y < p1.j; ++y) {
    real_t fy = abs((y - p_discrete.y) * filter_->inv_radius.y * PCF::filter_table_width);
    ifx[y - p0.j] = min((int) floor(fy), PCF::filter_table_width - 1);
  }
  for (auto ij : range2(p0, p1)) {
    // evaluate filter value at (x, y) pixel
    int offset = ify[ij.j - p0.j] * PCF::filter_table_width + ifx[ij.i - p0.i];
    real_t filterWeight = filter_->table[offset];
    // update pixel values with filtered sample contribution
    FilmTilePixel &pixel = getPixel(ij);
    pixel.contrib_sum += L * sample_weight * filterWeight;
    pixel.filter_weight_sum += filterWeight;
  }
  delete[] ifx;
  delete[] ify;
}

HERMES_DEVICE_CALLABLE FilmTilePixel &FilmTile::getPixel(const index2 &p) {
  int width = pixel_bounds_.upper().i - pixel_bounds_.lower().i;
  int offset = (p.i - pixel_bounds_.lower().i) + (p.j - pixel_bounds_.lower().j) * width;
  return pixels_[offset];
}

HERMES_DEVICE_CALLABLE const FilmTilePixel &FilmTile::getPixel(const index2 &p) const {
  int width = pixel_bounds_.upper().i - pixel_bounds_.lower().i;
  int offset = (p.i - pixel_bounds_.lower().i) + (p.j - pixel_bounds_.lower().j) * width;
  return pixels_[offset];
}

HERMES_DEVICE_CALLABLE bounds2i FilmTile::getPixelBounds() const { return pixel_bounds_; }

Film::Film(const hermes::size2 &resolution, Filter *filter, real_t diagonal, const bounds2 &crop_window)
    : full_resolution(resolution),
      cropped_pixel_bounds{bounds2i(
          index2(std::ceil(full_resolution.width * crop_window.lower.x),
                 std::ceil(full_resolution.height * crop_window.lower.y)),
          index2(std::ceil(full_resolution.width * crop_window.upper.x),
                 std::ceil(full_resolution.height * crop_window.upper.y)))},
      diagonal(diagonal * .001), filter(filter) {
}

HERMES_DEVICE_CALLABLE bounds2i Film::sampleBounds() const {
  return bounds2i(floor(point2(cropped_pixel_bounds.lower()) +
                      vec2(0.5) + filter.radius),
                  ceil(point2(cropped_pixel_bounds.upper()) +
                      vec2(0.5) - filter.radius));
}

HERMES_DEVICE_CALLABLE bounds2 Film::physicalExtent() const {
  real_t aspect = (real_t) full_resolution.height / (real_t) full_resolution.width;
  real_t x = std::sqrt(diagonal * diagonal / (1 + aspect * aspect));
  real_t y = aspect * x;
  return bounds2(point2(-x / 2, -y / 2), point2(x / 2, y / 2));
}

HERMES_DEVICE_CALLABLE FilmTile Film::filmTile(const bounds2i &sample_bounds) {
  // bound image pixels_ that samples in sample_bounds contribute to
  vec2 half_pixel = vec2(0.5f);
  bounds2 floatBounds = sample_bounds;
  index2 p0 = ceil(floatBounds.lower - half_pixel - filter.radius);
  index2 p1 = floor(floatBounds.upper - half_pixel + filter.radius) + vec2(1);
  bounds2i tilePixelBounds = intersect(bounds2i(p0, p1), cropped_pixel_bounds);
  return FilmTile(tilePixelBounds, &filter);
}

FilmImageView::FilmImageView(FilmPixel *pixels, f32 *rgb, Film film, real_t scale) : film_{std::move(film)},
                                                                                     pixels_{pixels},
                                                                                     rgb_{rgb}, scale{scale} {
}

HERMES_DEVICE_FUNCTION void FilmImageView::mergeFilmTile(const FilmTile &tile) {
  for (index2 pixel : tile.getPixelBounds()) {
    const FilmTilePixel &tilePixel = tile.getPixel(pixel);
    FilmPixel &mergePixel = getPixel(pixel);
    real_t xyz[3];
    tilePixel.contrib_sum.toXYZ(xyz);
    for (int i = 0; i < 3; ++i)
      atomicAdd(&mergePixel.xyz[i], xyz[i]);
    atomicAdd(&mergePixel.filter_weight_sum, tilePixel.filter_weight_sum);
  }
}

HERMES_DEVICE_CALLABLE FilmPixel &FilmImageView::getPixel(const index2 &p) {
  int width = film_.cropped_pixel_bounds.upper().i - film_.cropped_pixel_bounds.lower().i;
  int offset = (p.i - film_.cropped_pixel_bounds.lower().i) + (p.j - film_.cropped_pixel_bounds.lower().j) * width;
  return pixels_[offset];
}

HERMES_DEVICE_CALLABLE f32 *FilmImageView::rgb(const index2 &p) {
  int width = film_.cropped_pixel_bounds.upper().i - film_.cropped_pixel_bounds.lower().i;
  int offset = (p.i - film_.cropped_pixel_bounds.lower().i) + (p.j - film_.cropped_pixel_bounds.lower().j) * width;
  return &rgb_[offset * 3];
}

HERMES_DEVICE_FUNCTION Film &FilmImageView::film() {
  return film_;
}

FilmImage::FilmImage(const Film &film, real_t scale) : film_{film}, scale_{scale} {
  // allocate film image storage
  pixels_.resize(film.cropped_pixel_bounds.area());
  rgb_.resize(3 * film.cropped_pixel_bounds.area());
}

FilmImageView FilmImage::view() {
  return FilmImageView(pixels_.data(), rgb_.data(), film_, scale_);
}

const Film &FilmImage::film() const {
  return film_;
}

HERMES_CUDA_KERNEL(film2rgb)(FilmImageView film_image, real_t splat_scale = 1) {
  bounds2i bounds = film_image.film().cropped_pixel_bounds;
  HERMES_CUDA_THREAD_INDEX_IJ_LT(bounds.upper() - bounds.lower())
  auto p = ij + bounds.lower();
  // Pointer to RGB value
  real_t *rgb = film_image.rgb(p);
  // Convert pixel XYZ color to RGB
  FilmPixel &pixel = film_image.getPixel(p);
  XYZToRGB(pixel.xyz, rgb);
  // Normalize pixel with weight sum
  real_t filterWeightSum = pixel.filter_weight_sum;
  if (filterWeightSum != 0) {
    real_t invWt = (real_t) 1 / filterWeightSum;
    rgb[0] = max((real_t) 0, rgb[0] * invWt);
    rgb[1] = max((real_t) 0, rgb[1] * invWt);
    rgb[2] = max((real_t) 0, rgb[2] * invWt);
  }
  // Add splat value at pixel
  real_t splatRGB[3];
  real_t splatXYZ[3] = {pixel.splat_XYZ[0], pixel.splat_XYZ[1], pixel.splat_XYZ[2]};
  XYZToRGB(splatXYZ, splatRGB);

  rgb[0] += splat_scale * splatRGB[0];
  rgb[1] += splat_scale * splatRGB[1];
  rgb[2] += splat_scale * splatRGB[2];

  // Scale pixel value by _scale_
  rgb[0] *= film_image.scale;
  rgb[1] *= film_image.scale;
  rgb[2] *= film_image.scale;
}

Array<real_t> FilmImage::imagePixels() {
  f32 elapsed_time = 0;
  HERMES_CUDA_TIME(
      HERMES_CUDA_LAUNCH_AND_SYNC((pixels_.size()),
                                  film2rgb_k,
                                  view()),
      elapsed_time)
  HERMES_LOG_VARIABLE(elapsed_time)
  return rgb_;
}

//void FilmImageView::setImage(const SpectrumOld *img) const {
//  int nPixels = film_.cropped_pixel_bounds.area();
//  for (int i = 0; i < nPixels; ++i) {
//    Pixel &p = pixels_[i];
//    img[i].toXYZ(p.xyz);
//    p.filter_weight_sum = 1;
//    p.splat_XYZ[0] = p.splat_XYZ[1] = p.splat_XYZ[2] = 0;
//  }
//}

//void FilmImageView::addSplat(const point2 &p, const SpectrumOld &v) {
//  if (!film_.cropped_pixel_bounds.contains(p))
//    return;
//  real_t xyz[3];
//  v.toXYZ(xyz);
//  Pixel &pixel = getPixel(p);
//  for (int i = 0; i < 3; ++i)
//    pixel.splat_XYZ[i] += xyz[i]; // TODO ATOMIC!
//}

} // namespace helios